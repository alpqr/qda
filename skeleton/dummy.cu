#include "hip/hip_runtime.h"
#include "dummy.h"
#include <hip/hip_runtime.h>

__global__ void kernel(uchar4 *ptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    // ...
}

void Dummy::run()
{
    const int blockSize = 16;
    kernel<<<dim3(m_w / blockSize, m_h / blockSize), dim3(blockSize, blockSize)>>>((uchar4 *) m_devPtr);
}
